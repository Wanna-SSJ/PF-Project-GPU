#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include <iostream>
#include <random>
#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <chrono>
using namespace std;

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>

const int MaxT = 500;
const int MaxPart = 500000;
const int N = MaxPart;
const double sig_x = 2;
const double sig_obs = 0.5;

double randn() {
  static default_random_engine engine;
  static normal_distribution<double> dist(0, 1);
  return dist(engine);
}

struct trans {
  double a;
  trans(double a):a(a){};
  __host__ __device__
  double operator()(double x, double y) {
    return sin(a * x) + y;
  }
};

struct likeli {
  double y, coef;
  likeli(double y, double sig_obs):y(y),coef(-1.0/(2*sig_obs*sig_obs)){};
  __host__ __device__
  double operator()(double x) {
    // cpd y ~ N(x, sig_obs)
    //    ignore coefficient 1/(sigma * sqrt(2 PI))
    return exp(coef * (x - y) * (x - y));
  }
};

struct resample {
  double sum;
  double* W;
  double* X;
  int N;
  resample(double sum, double *W, double* X, int N)
    :sum(sum),W(W),X(X),N(N){};
  __host__ __device__
  double operator()(double t) {
    t = t * sum;
    int lo = 0, hi = N, mid;
    while(lo + 1 < hi) {
      mid = lo + hi >> 1;
      if (W[mid - 1] >= t) hi = mid;
      else lo = mid;
    }
    return X[hi - 1];
  }
};

int main() {

  double x, y, cur_x, a = -0.5;
  thrust::device_vector<double> X(N), T(N), Z(N), Y(N), W(N); 
  double* X_ptr = thrust::raw_pointer_cast(X.data());
  double* T_ptr = thrust::raw_pointer_cast(T.data());
  double* Z_ptr = thrust::raw_pointer_cast(Z.data());
  double* W_ptr = thrust::raw_pointer_cast(W.data()); 
  
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  int start, end;
  double duration;
  start = clock();
  // generate data
  thrust::host_vector<double> data(MaxT+1);
  for(int t=0; t<=MaxT; ++ t) {
    if (t == 0) cur_x = randn();
    else cur_x = sin(x * a) + randn() * sig_x;
    x = cur_x; y = cur_x + sig_obs * randn();
    data[t] = y;
  }
  end = clock();
  duration = (end - start) * 1.0 / CLOCKS_PER_SEC; 
  printf("Data Generation : %.6lfs\n", duration);


  start = clock();
  for(int t = 0; t <= MaxT; ++ t) {
    // fetch data
    y = data[t];
    // Particle Filter
    if (t == 0) {
      // initial value
      hiprandGenerateNormalDouble(gen, X_ptr, N, 0, 1);
    } else {
      // transition
      // x[t] = N(sin(a * x[t-1]), 2)
      hiprandGenerateNormalDouble(gen, Z_ptr, N, 0, 2);
      thrust::transform(X.begin(), X.end(), Z.begin(), X.begin(), trans(a));
    }
  
    // Calc Likelihood
    thrust::transform(X.begin(), X.end(), W.begin(), likeli(y, sig_obs));

    // resample
    thrust::inclusive_scan(W.begin(), W.end(), W.begin());

    double tot_sum = W[N - 1];
    hiprandGenerateUniformDouble(gen, Z_ptr, N);
    thrust::transform(Z.begin(), Z.end(), Y.begin(),
                      resample(tot_sum, W_ptr, X_ptr, N));
    thrust::copy(Y.begin(), Y.end(), X.begin());
  
  }

  end = clock();
  duration = 1.0 * (end - start) / CLOCKS_PER_SEC;
  printf("Time Elapsed: %.6lfds\n", duration);

}
