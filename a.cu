#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>
#include <hiprand.h>

int main() {
  int N = 6;

  thrust::host_vector<int> A(N);
  for(int i=0;i<N;++i)A[i]=i*i;

  thrust::device_vector<int> B = A;
  thrust::inclusive_scan(B.begin(), B.end(), B.begin());

  thrust::host_vector<int> C = B;
  for(int i = 0; i<A.size();++i) cout << A[i] << " "; cout << endl;
  for(int i=0;i<C.size();++i) cout << C[i] << " "; cout << endl;

  thrust::device_vector<double> D(N);
  double* D_ptr = thrust::raw_pointer_cast(D.data());
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniformDouble(gen, D_ptr, N);

  thrust::host_vector<double> E(N);
  thrust::copy(D.begin(), D.end(), E.begin());

  cout << "Random Gaussian : " << endl;
  for(int i = 0; i < E.size(); i++) cout << "  >> " << E[i] << endl;  

}
